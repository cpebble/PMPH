#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 256


__global__ void kernel(float *d_in, float *d_out, int N){
  const unsigned int lid = threadIdx.x; // Local id inside a block
  const unsigned int gid = blockIdx.x*blockDim.x + lid; // global id
  if (gid < N){
    d_out[gid] = powf(d_in[gid]/(d_in[gid]-2.3), 3);
  }
}

void gpu_run(float* inp, float* out, int N)
{
  // Most of this code is stolened from the lab1 slides
  unsigned int block_size = BLOCK_SIZE;
  unsigned int num_blocks = ((N + (block_size - 1)) / block_size);
  float* d_in;
  float* d_out;
  unsigned_int mem_size = N*sizeof(float);
  // Cuda pointers calculated behind the scenes
  hipMalloc((void**)&d_in, mem_size);
  hipMalloc((void**)&d_out, mem_size);
  // Copy host mem to device
  hipMemcpy(d_in, inp, mem_size, hipMemcpyHostToDevice);
  // Exec kernel
  kernel<<<num_blocks, block_size>>>(d_in, d_out, N);
  // Copy result from device to host
  hipMemcpy(out, d_out, mem_size, hipMemcpyDeviceToHost);
  hipFree(d_in); hipFree(d_out);
}

int main( int argc, char** argv){
  unsigned int N = 753411;
  unsigned int mem_size = N*sizeof(float);
  // Init memory arrays
  float* in = (float*) malloc(mem_size);
  float* gpu_out = (float*) malloc(mem_size);
  float* seq_out = (float*) malloc(mem_size);
  // And init the input array
  for (unsigned int i=0; i<N; ++i) h_in[i] = (float)i;

  // Run the code on the GPU
  gpu_run(in, gpu_out, N);
  // Free outpus databases
  free(in); free(gpu_out); free(seq_out);

  return 0;
}
