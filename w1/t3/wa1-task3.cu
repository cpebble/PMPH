#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <time.h>
#ifndef BLOCK_SIZE
  #define BLOCK_SIZE 256
#endif
#ifndef N_ELEMS
  #define N_ELEMS 753411
#endif


// Src: Lab1-CudaIntro. Get time difference
int timeval_subtract( 
        struct timeval *result,
        struct timeval *t2,
        struct timeval *t1)
{
  unsigned int resolution = 1000000;
  long int diff = (t2->tv_usec + resolution * t2->tv_sec) - 
                  (t1->tv_usec + resolution * t2->tv_sec);
  result->tv_sec = diff / resolution; result->tv_usec = diff % resolution;
  return (diff<0);
}

__global__ void kernel(float *d_in, float *d_out, int N){
  const unsigned int lid = threadIdx.x; // Local id inside a block
  const unsigned int gid = blockIdx.x*blockDim.x + lid; // global id
  if (gid < N){
    d_out[gid] = powf(d_in[gid]/(d_in[gid]-2.3), 3);
  }
}

void gpu_run(float* inp, float* out, int N)
{
  // Most of this code is stolened from the lab1 slides
  // Time tracking vars
  unsigned long int elapsed; 
  struct timeval t_start, t_end, t_diff;

  // Block distr vars
  unsigned int block_size = BLOCK_SIZE;
  unsigned int num_blocks = ((N + (block_size - 1)) / block_size);

  // Memory assignment
  unsigned int mem_size = N*sizeof(float);
  float* d_in;
  float* d_out;
  hipMalloc((void**)&d_in, mem_size);
  hipMalloc((void**)&d_out, mem_size);

  // Copy host mem to device
  hipMemcpy(d_in, inp, mem_size, hipMemcpyHostToDevice);
  // Exec kernel(with timetrack)
  gettimeofday(&t_start, NULL);
  kernel<<<num_blocks, block_size>>>(d_in, d_out, N);
  gettimeofday(&t_end, NULL);
  // Copy result from device to host
  hipMemcpy(out, d_out, mem_size, hipMemcpyDeviceToHost);
  hipFree(d_in); hipFree(d_out);
  // Calculate and print time
  timeval_subtract(&t_diff, &t_end, &t_start);
  elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);
  printf("GPU Run took %d microseconds (%.2fms)\n", elapsed, elapsed / 1000.0);
}

void seq_run(float* inp, float* out, int N){
  unsigned long int elapsed; 
  struct timeval t_start, t_end, t_diff;
  gettimeofday(&t_start, NULL);
  for(unsigned int i = 0; i < N; ++i){
    out[i] = pow(inp[i]/(inp[i]-2.3), 3);
  }
  gettimeofday(&t_end, NULL);
  timeval_subtract(&t_diff, &t_end, &t_start);
  elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);
  printf("CPU Run took %d microseconds (%.2fms)\n", elapsed, elapsed / 1000.0);
}

int main( int argc, char** argv){
  unsigned int N = N_ELEMS;
  unsigned int mem_size = N*sizeof(float);
  // Init memory arrays
  float* in = (float*) malloc(mem_size);
  float* gpu_out = (float*) malloc(mem_size);
  float* seq_out = (float*) malloc(mem_size);
  // And init the input array
  for (unsigned int i=0; i<N; ++i) in[i] = (float)i;

  // Run the code on the CPU
  seq_run(in, seq_out, N);
  // Run the code on the GPU
  gpu_run(in, gpu_out, N);

  // Now validate results:
  int passed = 0;
  int invalid = 0;
  for (int i = 0; i < N; ++i) {
    if (fabs(seq_out[i] - gpu_out[i]) < 0.0001)
        passed++;
    else invalid++;
  }
  printf("Passed: %06d, Invalid: %06d\n", passed, invalid);

  //DEBUG: Print the first 10 and last 10 values to 10p of precision
  // for(int i = 0; i < 10; i++) printf("%6d:\t%.10f\t%.10f\n", i, seq_out[i], gpu_out[i]);
  // for(int i = 0; i < 10; i++) printf("%6d:\t%.10f\t%.10f\n", N-i, seq_out[N-i], gpu_out[N-i]);
  // Free outpus databases
  free(in); free(gpu_out); free(seq_out);

  return 0;
}
